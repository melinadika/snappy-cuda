#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <limits.h>
#include <getopt.h>
#include <string>
#include <iostream>

#include "snappy_cuda.h"
#include "snappy_compress.h"
#include "snappy_decompress.h"



/**
 * Read the contents of a file into an in-memory buffer. Upon success,
 * writes the amount read to input->length.
 *
 * @param in_file: input file name.
 * @param input: holds input buffer information
 * @return 1 if file does not exist, is too long, or different number of bytes
 *         were read than expected, 0 otherwise
 */
static int read_input_host(char *in_file, struct host_buffer_context *input)
{
	FILE *fin = fopen(in_file, "r");
	if (fin == NULL) {
		fprintf(stderr, "Invalid input file: %s\n", in_file);
		return 1;
	}

	fseek(fin, 0, SEEK_END);
	input->length = ftell(fin);
	fseek(fin, 0, SEEK_SET);

	if (input->length > input->max) {
		fprintf(stderr, "input_size is too big (%ld > %ld)\n",
				input->length, input->max);
		return 1;
	}

	input->buffer = (uint8_t *)malloc(ALIGN_LONG(input->length, 8) * sizeof(*(input->buffer)));
	input->curr = input->buffer;
	size_t n = fread(input->buffer, sizeof(*(input->buffer)), input->length, fin);
	fclose(fin);

#ifdef DEBUG
	printf("%s: read %ld bytes from %s (%lu)\n", __func__, input->length, in_file, n);
#endif
   return (n != input->length);
}

/**
 * Read the contents of a file into an in-memory buffer. Upon success,
 * writes the amount read to input->length.
 *
 * @param in_file: input file name.
 * @param input: holds input buffer information
 * @return 1 if file does not exist, is too long, or different number of bytes
 *         were read than expected, 0 otherwise
 */
static int read_input_cuda(char *in_file, struct host_buffer_context *input)
{
	FILE *fin = fopen(in_file, "r");
	if (fin == NULL) {
		fprintf(stderr, "Invalid input file: %s\n", in_file);
		return 1;
	}

	fseek(fin, 0, SEEK_END);
	input->length = ftell(fin);
	fseek(fin, 0, SEEK_SET);

	if (input->length > input->max) {
		fprintf(stderr, "input_size is too big (%ld > %ld)\n",
				input->length, input->max);
		return 1;
	}

	//input->buffer = (uint8_t *)malloc(ALIGN_LONG(input->length, 8) * sizeof(*(input->buffer)));
	input->total_size = ALIGN_LONG(input->length, 8) * sizeof(*(input->buffer));
	checkCudaErrors(hipMallocManaged(&input->buffer,input->total_size));
	input->curr = input->buffer;
	size_t n = fread(input->buffer, sizeof(*(input->buffer)), input->length, fin);
	fclose(fin);

#ifdef DEBUG
	printf("%s: read %ld bytes from %s (%lu)\n", __func__, input->length, in_file, n);
#endif
   return (n != input->length);
}

/**
 * Write the contents of the output buffer to a file.
 *
 * @param out_file: output filename.
 * @param output: holds output buffer information
 */
static void write_output_host(char *out_file, struct host_buffer_context *output)
{
	FILE *fout = fopen(out_file, "w");
	fwrite(output->buffer, 1, output->length, fout);
	fclose(fout);
}

/**
 * Print out application usage.
 *
 * @param exe_name: name of the application
 */
static void usage(const char *exe_name)
{
#ifdef DEBUG
	fprintf(stderr, "**DEBUG BUILD**\n");
#endif //DEBUG
	fprintf(stderr, "Compress or decompress a file with Snappy\nCan use either the host CPU or CUDA\n");
	fprintf(stderr, "usage: %s [-d] [-x <cuda blocks>] [-y <cuda threads per block] [-c] [-b <block_size>] -i <input_file> [-o <output_file>]\n", exe_name);
	fprintf(stderr, "d: use CUDA, by default host is used\n");
	fprintf(stderr, "x: Grid size - number of blocks (Carefull! no error checks are done)\n");
	fprintf(stderr, "y: number of threads per block (Carefull! no error checks are done)\n");
	fprintf(stderr, "c: perform compression, by default performs decompression\n");
	fprintf(stderr, "b: block size used for compression, default is 32KB, ignored for decompression\n");
	fprintf(stderr, "i: input file\n");
	fprintf(stderr, "o: output file\n");
}

/**
 * Calculate the difference between two timeval structs.
 */
double get_runtime(struct timeval *start, struct timeval *end) {
	double start_time = start->tv_sec + start->tv_usec / 1000000.0;
	double end_time = end->tv_sec + end->tv_usec / 1000000.0;
	return (end_time - start_time);
}

int main(int argc, char **argv)
{
	int opt;
	snappy_status status;
	
	int compress = 0;
	int block_size = 32 * 1024; // Default is 32KB
    char * input_file = NULL;
    char * output_file = NULL;
    const char * default_output_file = "output.txt";
	struct host_buffer_context *input;
	struct host_buffer_context *output;
	struct program_runtime runtime;

	// use defaults which are set later
	memset(&runtime, 0, sizeof(runtime));
	input_file = argv[1];

	checkCudaErrors(hipMallocManaged(&input,sizeof(host_buffer_context)));
	checkCudaErrors(hipMallocManaged(&output,sizeof(host_buffer_context)));

	input->buffer = NULL;
	input->length = 0;
	input->max = ULONG_MAX;

	output->buffer = NULL;
	output->length = 0;
	output->max = ULONG_MAX;

	if (!input_file)
	{
		usage(argv[0]);
		return -1;
	}
	input->file_name = input_file;

	// Read the input file into main memory

	if (read_input_cuda(input_file, input))
		return -1;
	input->block_size = block_size;
	

	if (setup_decompression_cuda(input, output, &runtime))
		return -1;


	status = snappy_decompress_cuda(input, output, &runtime);
	
	
	if (status == SNAPPY_OK)
	{

		terminate_decompression(input, output, &runtime);
			
	}
	else
	{
		fprintf(stderr, "Encountered Snappy error %u\n", status);
		return -1;
	}


	checkCudaErrors(hipFree(input));
	checkCudaErrors(hipFree(output));
	
	
	return 0;
}

